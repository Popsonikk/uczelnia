#include <hip/hip_runtime.h>
#include <iostream>


__global__ void addKernel(int * a ,int * b,int * res  )
{
    int index=threadIdx.x;
    res[index]=a[index]+b[index];
    
}

int main() {
    const int size=512;
    int *a=(int*)malloc(size*sizeof(int));
    int *b=(int*)malloc(size*sizeof(int));
    int *res=(int*)malloc(size*sizeof(int));
    for(int i=0;i<size;i++)
    {
        a[i]=i;
        b[i]=i;
    }
    int* a_gpu;
    int* b_gpu;
    int* res_gpu;
    hipMalloc(&a_gpu,sizeof(int)*size);
    hipMalloc(&b_gpu,sizeof(int)*size);
    hipMalloc(&res_gpu,sizeof(int)*size);
    hipMemcpy(a_gpu, a,sizeof(int)*size,hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b,sizeof(int)*size,hipMemcpyHostToDevice);

    addKernel <<<1,size>>> (a_gpu,b_gpu,res_gpu);
    hipDeviceSynchronize();
    
    hipMemcpy(res, res_gpu,sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<size;i++)
        std::cout<<res[i]<<"|";

    return 0;
}