#include <hip/hip_runtime.h>
#include <iostream>


__global__ void initKernel(long * tab) {
    int index=threadIdx.x+blockIdx.x*blockDim.x;
    tab[index]=index;
    
    
}
__global__ void reduction(long * tab ,int step)
{
    int index=threadIdx.x+blockIdx.x*blockDim.x;
    if(index<step)
    tab[index]+=tab[index+step];
    
}

int main() {
    const int size=512;
    const int numblock=16;
    long * tab_gpu;
    hipMalloc(&tab_gpu,sizeof(long)*size*numblock);
    initKernel <<<numblock,size>>> (tab_gpu);
    hipDeviceSynchronize();
    for(int step=size*numblock;step>=1;step/=2)
    {
        reduction <<<numblock,size>>>(tab_gpu,step);
    }
    int result;
    hipMemcpy(&result, tab_gpu,sizeof(long),hipMemcpyDeviceToHost);
    std::cout<<result<<std::endl;

    return 0;
}